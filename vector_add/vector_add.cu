#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void sumArrays(int *A, int *B, long long *partialSum, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < size) {
            partialSum[idx] = A[idx] + B[idx];
        }
}

__global__ void reduce(long long *partialSum, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = 1;

    while (stride < size) {
        if (idx < size - stride) {
            partialSum[idx] += partialSum[idx + stride];
        }
        __syncthreads();
        stride *= 2;
    }
}

int main () {
    cout << "Program Start" << endl;
    long long size = 1000000;
    long long sum = 0;

    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;

    vector<int> A(size), B(size);
    vector <long long> partialSum(size);

    for (int i = 0; i < size; i++) {
        A[i] = i;
        B[i] = size - i;
    }

    //Pointers to GPU and memory allocation
    int *d_A, *d_B;
    long long *d_partialSum;
    hipMalloc(&d_A, size * sizeof(int));
    hipMalloc(&d_B, size * sizeof(int));
    hipMalloc(&d_partialSum, numBlocks * sizeof(long long));

    //Copy A and B from CPU -> GPU
    hipMemcpy(d_A, A.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), size * sizeof(int), hipMemcpyHostToDevice);

    sumArrays<<<numBlocks, blockSize>>>(d_A, d_B, d_partialSum, size);

    reduce<<<numBlocks, blockSize>>>(d_partialSum, size);

    hipMemcpy(&sum, d_partialSum, sizeof(long long), hipMemcpyDeviceToHost);

    cout << "Sum: " << sum << endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_partialSum);

    return 0;
}
