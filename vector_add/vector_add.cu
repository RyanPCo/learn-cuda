#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void sumArrays(int *A, int *B, long long *partialSum, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < size) {
            partialSum[idx] = A[idx] + B[idx];
        }
}

__global__ void reduce(long long *partialSum, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = 1;

    while (stride < size) {
        if (idx < size - stride) {
            partialSum[idx] += partialSum[idx + stride];
        }
        __syncthreads();
        stride *= 2;
    }
}

int main () {
    cout << "Program Start" << endl;
    long long size = 1000000000;
    int sum = 0;

    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;

    vector<int> A(size), B(size);
    vector <long long> partialSum(size);

    for (int i = 0; i < size; i++) {
        A[i] = i;
        B[i] = B.size() - i;
    }

    int *d_A, *d_B;
    long long *d_partialSum;
    hipMalloc(&d_A, size * sizeof(int));
    hipMalloc(&d_B, size * sizeof(int));
    hipMalloc(&d_partialSum, numBlocks * sizeof(long long));

    hipMemcpy(d_A, A.data(), size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), size * sizeof(int), hipMemcpyHostToDevice);

    cout << "Copy Memory" << endl;

    sumArrays<<<numBlocks, blockSize>>>(d_A, d_B, d_partialSum, size);

    cout << "Sum" << endl;

    reduce<<<numBlocks, blockSize>>>(d_partialSum, size);

    cout << "Reduce" << endl;

    hipMemcpy(&sum, d_partialSum, sizeof(long long), hipMemcpyDeviceToHost);

    cout << sum << endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_partialSum);

    return 0;
}
